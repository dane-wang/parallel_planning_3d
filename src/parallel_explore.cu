#include "hip/hip_runtime.h"


#include <time.h>  
#include <vector>
#include "parallel_planning_3d/planner.h"
#include "parallel_planning_3d/parallel_explore.cuh"
#include "std_msgs/Int32MultiArray.h"
#include <algorithm>
#include <xmlrpcpp/XmlRpcValue.h>

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/merge.h>
#include <queue>

__device__ bool path_found_gpu;
__device__ int neighbor_gpu[3*26];
__device__ int goal_gpu;

struct is_negative
{
  __host__ __device__
  bool operator()(int x)
  {
    return x ==-1;
  }
};



template <typename T, typename T1> 
__global__ void get_f(T* q,  planner::Node* graph, T1* h, int q_size )
{

  int tid = blockIdx.x *blockDim.x + threadIdx.x;
  if (tid < q_size){
    int node = q[tid];

    h[tid] = graph[node].f;

    // printf("%d", q[tid]);
  }

}

template <typename T>
__global__ void explore(T* q,  planner::Node* graph, T* new_q, int q_size, int n  )
{
  int tid = blockIdx.x *blockDim.x + threadIdx.x;
  if (tid<q_size) {

    int explored_index = q[tid];
    
    int explored_coord[3];
    explored_coord[2] = explored_index/(n*n);

    int a = explored_index%(n*n);

    explored_coord[0] = a%n;
    explored_coord[1] = a/n;

    graph[explored_index].explored = true;
    graph[explored_index].frontier = false;

    if (graph[explored_index].goal){
      printf("FOUND");
      printf("Hello from thread %d, I am exploring %d\n", tid, explored_index);
      // planner::Node* temp_node = graph[explored_index].parent;
      // while (!temp_node->start){
        
      //     temp_node->path = true;
      //     temp_node = temp_node->parent;
      // }
      path_found_gpu = true;
    }

    if (!path_found_gpu){
      for (int i=0; i<26; i++)
      {   
        
        int neighbor[3];
        neighbor[0] = neighbor_gpu[3*i];
        neighbor[1] = neighbor_gpu[3*i+1];
        neighbor[2] = neighbor_gpu[3*i+2];  

        int new_coord[3];
        new_coord[0] = explored_coord[0] + neighbor[0];
        new_coord[1] = explored_coord[1] + neighbor[1];
        new_coord[2] = explored_coord[2] + neighbor[2];

        // printf("Checking %d, %d, %d\n", (int) new_coord[0], (int) new_coord[1], (int) new_coord[2]);

        int new_index = new_coord[0] + new_coord[1]*n + new_coord[2]*n*n;
        
        
        if (new_index<0 || new_index >= n*n*n) continue;

        float cost;
        
        if (i<6){
          cost = 1;
        }
        else if (i<18)
        {
          cost = sqrt(2.0);
        }
        else {
          cost = sqrt(3.0);
        }

        bool edge_detect = true;

        
        if ((new_coord[0] >= n) || (new_coord[0] < 0)  || (new_coord[1] >= n) || (new_coord[1] <0 ) || (new_coord[2] >= n) || (new_coord[2] < 0)){
            edge_detect = false;
        }

        if (graph[new_index].obstacle == false && graph[new_index].frontier == false && graph[new_index].explored == false && edge_detect)
        {
          graph[new_index].g = graph[explored_index].g + cost;
            
          float h_1 = sqrt(pow((graph[new_index].x-graph[goal_gpu].x),2) + pow((graph[new_index].y-graph[goal_gpu].y),2) + pow((graph[new_index].z-graph[goal_gpu].z),2) );
            // printf("%f", h_1);
          graph[new_index].h = h_1;

            
          graph[new_index].f = graph[new_index].h + graph[new_index].g;
          graph[new_index].parent = explored_index;
          graph[new_index].frontier = true;
          
          new_q[26*tid+i] = new_index;
        }
        else if (edge_detect && graph[new_index].obstacle == false && (graph[new_index].frontier == true || graph[new_index].explored == true))
        {
          if (graph[new_index].g > graph[explored_index].g + cost)
          {
            graph[new_index].g = graph[explored_index].g + cost;
            graph[new_index].f = graph[new_index].h + graph[new_index].g;
            graph[new_index].parent = explored_index;
          }
        }
      }

    }
  }

}


__global__ void warmup(int* a)
{
    a = a+1;
}


extern "C"
void parallel_explore(planner::Node* graph, int n, int start_index, int goal_index, int max_thread, std::vector<int>& path_to_goal){

  //Setup everything for planning
  // graph[start_index].g = 0;
  // graph[start_index].h = h_calculation(&graph[start_index], &graph[goal_index]);
  // graph[start_index].f = graph[start_index].g + graph[start_index].h;
  bool path_found = false;
  int goal = goal_index;
  thrust::host_vector<int> q_lists;
  q_lists.push_back(start_index);

  const int map_size = n*n*n*sizeof(planner::Node);

  planner::Node *map_gpu;

  int neighbors[][3] = {{0, 0, 1}, {0, 0, -1}, {0, 1, 0}, {0, -1, 0}, {1, 0, 0}, {-1, 0, 0}, {0, 1, 1}, {0, 1, -1}, {0, -1, 1}, {0, -1, -1}, {1, 0, 1}, {1, 0, -1}, {1, 1, 0}, {1, -1, 0}, {-1, 0, 1} , {-1, 0, -1} , {-1, 1, 0} , {-1, -1, 0} , {1, 1, 1} , {1, 1, -1} , {1, -1, 1} , {1, -1, -1} , {-1, -1, -1} , {-1, -1, 1} , {-1, 1, -1} , {-1, 1, 1}   };

  int neighbor[26*3];
  for (int i =0; i< 26; i++){
    for (int j=0; j<3; j++){

      neighbor[3*i+j] = neighbors[i][j];

    }

  }
  //Copy all needed variables to gpu
  hipMalloc( (void**)&map_gpu, map_size );
  hipMemcpy(map_gpu, graph, map_size, hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(path_found_gpu), &path_found,  sizeof(bool));
  hipMemcpyToSymbol(HIP_SYMBOL(neighbor_gpu), &neighbor,  3*26*sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(goal_gpu), &goal,  sizeof(int));

  //q list on gpu
  thrust::device_vector<int> q_lists_gpu = q_lists;

  while(q_lists_gpu.size()!=0 && !path_found){
    int q_size = q_lists_gpu.size();

    //Determine how many thread should be launched
    int thread_size_needed = min(max_thread, q_size);
    int block_size, thread_size;

    if (thread_size_needed <=1024){
      block_size = 1;
      thread_size = thread_size_needed;
    }
    else{
      block_size = (thread_size_needed/1024) + 1;
      thread_size = 1024;
    }



    //new_q is the list store the frontier generated from this step of exploration
    thrust::device_vector<int> new_q_lists_gpu(26*thread_size_needed);
    thrust::fill(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), -1);


    


    //Launch the kernel to explore the map
    explore<<<block_size,thread_size>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(new_q_lists_gpu.data()), thread_size_needed, n);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&path_found, HIP_SYMBOL(path_found_gpu),  sizeof(bool), 0, hipMemcpyDeviceToHost );
    // hipMemcpy(&graph, map_gpu,  map_size, hipMemcpyDeviceToHost );


    // Remove all element that is not used during the exploration and repeated value
    
    new_q_lists_gpu.erase(thrust::remove_if(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), is_negative()),  new_q_lists_gpu.end() );
    thrust::sort(new_q_lists_gpu.begin(), new_q_lists_gpu.end());
    new_q_lists_gpu.erase(thrust::unique(new_q_lists_gpu.begin(), new_q_lists_gpu.end()), new_q_lists_gpu.end() );
    
    
    // Create new q list based on origional and updated q
    if (q_size <= max_thread) {
      q_lists_gpu.clear();
      q_lists_gpu = new_q_lists_gpu;
      new_q_lists_gpu.clear();
    }
    else {
      
      q_lists_gpu.erase(q_lists_gpu.begin(), q_lists_gpu.begin()+max_thread );
      q_lists_gpu.insert(q_lists_gpu.end(), new_q_lists_gpu.begin(), new_q_lists_gpu.end() );
      thrust::sort(q_lists_gpu.begin(), q_lists_gpu.end());
      q_lists_gpu.erase(thrust::unique(q_lists_gpu.begin(), q_lists_gpu.end()), q_lists_gpu.end() );
    
      new_q_lists_gpu.clear();

      // //sort the q_list based on the f value
      thrust::device_vector<float> f_value(q_lists_gpu.size());
      get_f<<<1, q_lists_gpu.size()>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(f_value.data()), q_lists_gpu.size() );
      hipDeviceSynchronize();
      thrust::sort_by_key(f_value.begin(), f_value.end(), q_lists_gpu.begin() );
      
    }

    
    //q_size = q_lists_gpu.size();
    // thrust::device_vector<float> h_value(q_size);

    // if (q_size > 1024) {
    //   int block = q_size / 1024 + 1;
      
    //   get_h<<<block, 1024>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(&h_value[0]), q_size );

    //   thrust::sort_by_key(h_value.begin(), h_value.end(), q_lists_gpu.begin() );

    // }

    if (path_found){
      hipMemcpy(graph, map_gpu,  map_size, hipMemcpyDeviceToHost );
      int path1 = goal;
      while (path1 != start_index)
        {  
          path_to_goal.push_back(path1);
          graph[path1].path = true;
          // path.push_back(path1);
          path1 = graph[path1].parent;
        }
      // hipFree(map_gpu);


    }
  }

  if (q_lists_gpu.size()==0) std::cout<< "NO PATH IS FOUND" <<std::endl;        
    

 
}

extern "C"
void gpu_warmup() {

    //GPU warm up
    int a = 0;
    int* a_gpu;
    hipMalloc( (void**)&a_gpu, sizeof(int) );
    hipMemcpy(a_gpu, &a, sizeof(int), hipMemcpyHostToDevice);
    warmup<<<1,1>>>(a_gpu);
    hipDeviceSynchronize();
}

#include "hip/hip_runtime.h"
#include <ros/ros.h> 
//#include <geometry_msgs>
// #include "geometry_msgs/Point.h"
#include <time.h>  
#include <vector>
#include "parallel_planning_3d/planner.h"
#include "std_msgs/Int32MultiArray.h"
#include <algorithm>
#include <xmlrpcpp/XmlRpcValue.h>

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/merge.h>
#include <queue>
#include <random>


__device__ bool path_found_gpu;
__device__ int neighbor_gpu[3*26];
__device__ int goal_gpu;


struct is_negative
{
  __host__ __device__
  bool operator()(int x)
  {
    return x ==-1;
  }
};



template <typename T, typename T1> 
__global__ void get_f(T* q,  planner::Node* graph, T1* h, int q_size )
{

  int tid = blockIdx.x *blockDim.x + threadIdx.x;
  if (tid < q_size){
    int node = q[tid];

    h[tid] = graph[node].f;

    // printf("%d", q[tid]);
  }

}

template <typename T>
__global__ void explore(T* q,  planner::Node* graph, T* new_q, int q_size , int n )
{
  int tid = blockIdx.x *blockDim.x + threadIdx.x;
  if (tid<q_size) {

    int explored_index = q[tid];
  
    int explored_coord[3];
    explored_coord[2] = explored_index/(n*n);

    int a = explored_index%(n*n);

    explored_coord[0] = a%n;
    explored_coord[1] = a/n;


    graph[explored_index].explored = true;
    graph[explored_index].frontier = false;

    if (graph[explored_index].goal){
      printf("FOUND");
      printf("Hello from thread %d, I am exploring %d\n", tid, explored_index);
      // planner::Node* temp_node = graph[explored_index].parent;
      // while (!temp_node->start){
        
      //     temp_node->path = true;
      //     temp_node = temp_node->parent;
      // }
      path_found_gpu = true;
    }

    if (!path_found_gpu){
      for (int i=0; i<26; i++)

      { 

         
        int neighbor[3];
        neighbor[0] = neighbor_gpu[3*i];
        neighbor[1] = neighbor_gpu[3*i+1];
        neighbor[2] = neighbor_gpu[3*i+2];  

        int new_coord[3];
        new_coord[0] = explored_coord[0] + neighbor[0];
        new_coord[1] = explored_coord[1] + neighbor[1];
        new_coord[2] = explored_coord[2] + neighbor[2];

        // printf("Checking %d, %d, %d\n", (int) new_coord[0], (int) new_coord[1], (int) new_coord[2]);

        int new_index = new_coord[0] + new_coord[1]*n + new_coord[2]*n*n;
        
        
        
        
        
        if (new_index<0 || new_index >= n*n*n) continue;

        float cost;
        
        if (i<6){
          cost = 1;
        }
        else if (i<18)
        {
          cost = sqrt(2.0);
        }
        else {
          cost = sqrt(3.0);
        }

        bool edge_detect = true;
     
        if ((new_coord[0] >= n) || (new_coord[0] < 0)  || (new_coord[1] >= n) || (new_coord[1] <0 ) || (new_coord[2] >= n) || (new_coord[2] < 0)){
            edge_detect = false;
        }

        


        if (graph[new_index].obstacle == false && graph[new_index].frontier == false && graph[new_index].explored == false && edge_detect)
        {
          graph[new_index].g = graph[explored_index].g + cost;
            
          float h_1 = sqrt(pow((graph[new_index].x-graph[goal_gpu].x),2) + pow((graph[new_index].y-graph[goal_gpu].y),2) + pow((graph[new_index].z-graph[goal_gpu].z),2) );
            // printf("%f", h_1);
          graph[new_index].h = h_1;

            
          graph[new_index].f = graph[new_index].h + graph[new_index].g;
          graph[new_index].parent = explored_index;
          graph[new_index].frontier = true;
          
          new_q[26*tid+i] = new_index;
        }
        else if (edge_detect && graph[new_index].obstacle == false && (graph[new_index].frontier == true || graph[new_index].explored == true))
        {
          if (graph[new_index].g > graph[explored_index].g + cost)
          {
            graph[new_index].g = graph[explored_index].g + cost;
            graph[new_index].f = graph[new_index].h + graph[new_index].g;
            graph[new_index].parent = explored_index;
          }
        }
      }

    }
  }

}




  


int main(int argc, char** argv)
{
  ros::init(argc, argv, "parallel_planning");
  ros::NodeHandle nh; 

  // 发布消息 话题名字 队列大小
  ros::Publisher pub = nh.advertise<std_msgs::Int32MultiArray> ("planning_info", 100, ros::init_options::AnonymousName);
    
  //geometry_msgs::Point start_goal;
  std_msgs::Int32MultiArray map;
  

  //generate map info from the config file
  int n, max_thread_size, use_random_obstacles;
  std::vector<int> start_coord, goal_coord;
  std::vector<int> obstacles;
 

  ros::param::get("map_size", n);
  ros::param::get("start_position", start_coord);
  ros::param::get("goal_position", goal_coord);
  ros::param::get("use_random_obstacles", use_random_obstacles);
  ros::param::get("max_thread", max_thread_size);

  // Initialize the start and goal node
  int start = start_coord[0]+ start_coord[1] * n + start_coord[2] * n * n;
  int goal = goal_coord[0] + goal_coord[1] * n + goal_coord[2] * n * n;

  if(use_random_obstacles){

        float ratio;
        ros::param::get("random_obstacles_ratio", ratio);

        int obstacle_size = ratio * n;

        // std::cout<<"obstacle "<< obstacle_size<< std::endl;

        // First create an instance of an engine.
        std::random_device rnd_device;
        // Specify the engine and distribution.
        std::mt19937 mersenne_engine {rnd_device()};  // Generates random integers
        std::uniform_int_distribution<int> dist {0, n*n*n-1};
        
        auto gen = [&dist, &mersenne_engine](){
                    return dist(mersenne_engine);
                };

        
        std::vector<int> vec(obstacle_size);
        std::generate(std::begin(vec), std::end(vec), gen);
        obstacles = vec;

        

    }
    else{

        XmlRpc::XmlRpcValue xml_obstacles;
        ros::param::get("obstacles", xml_obstacles);
        for(int i=0; i< xml_obstacles.size(); i++){
            int obstacles_index =  (int)xml_obstacles[i][0] +  (int)xml_obstacles[i][1] * n + (int)xml_obstacles[i][2] * n * n;
            obstacles.push_back(obstacles_index);
        }
  }


  planner::Node* graph = new planner::Node[n*n*n];

	planner::map_generation(graph, n, start, goal, obstacles);

  int path1 = goal;
  bool path_found = false;

  // Start to work with CUDA
  thrust::host_vector<int> q_lists;

  q_lists.push_back(start);

  // Start to allocate memory on gpu:
  
  const int map_size = n*n*n*sizeof(planner::Node);

  planner::Node *map_gpu;

  int neighbors[][3] = {{0, 0, 1}, {0, 0, -1}, {0, 1, 0}, {0, -1, 0}, {1, 0, 0}, {-1, 0, 0}, {0, 1, 1}, {0, 1, -1}, {0, -1, 1}, {0, -1, -1}, {1, 0, 1}, {1, 0, -1}, {1, 1, 0}, {1, -1, 0}, {-1, 0, 1} , {-1, 0, -1} , {-1, 1, 0} , {-1, -1, 0} , {1, 1, 1} , {1, 1, -1} , {1, -1, 1} , {1, -1, -1} , {-1, -1, -1} , {-1, -1, 1} , {-1, 1, -1} , {-1, 1, 1}   };

  int neighbor[26*3];
  for (int i =0; i< 26; i++){
    for (int j=0; j<3; j++){

      neighbor[3*i+j] = neighbors[i][j];



    }

  }

  hipMalloc( (void**)&map_gpu, map_size );
  hipMemcpy(map_gpu, graph, map_size, hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(path_found_gpu), &path_found,  sizeof(bool));
  hipMemcpyToSymbol(HIP_SYMBOL(neighbor_gpu), &neighbor,  3*26*sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(goal_gpu), &goal,  sizeof(int));


  thrust::device_vector<int> q_lists_gpu = q_lists;

  // parallel_explore(&graph[0], n, path_found, start, max_thread_size);
  
  


  while (ros::ok()) {
    while(ros::ok() && q_lists_gpu.size()!=0 && !path_found){

      int q_size = q_lists_gpu.size();
      // std::cout << "q size is" << q_size << std::endl;

      
      

      //Determine how many thread should be launched
      int thread_size_needed = min(max_thread_size, q_size);
      int block_size, thread_size;

      if (thread_size_needed <=1024){
        block_size = 1;
        thread_size = thread_size_needed;
      }
      else{
        block_size = (thread_size_needed/1024) + 1;
        thread_size = 1024;
      }

      //new_q is the list store the frontier generated from this step of exploration
      thrust::device_vector<int> new_q_lists_gpu(26*thread_size_needed);
      thrust::fill(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), -1);


 
      //Launch the kernel to explore the map
      explore<<<block_size,thread_size>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(new_q_lists_gpu.data()), thread_size_needed, n);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&path_found, HIP_SYMBOL(path_found_gpu),  sizeof(bool), 0, hipMemcpyDeviceToHost );

      
      hipMemcpy(graph, map_gpu,  map_size, hipMemcpyDeviceToHost );


      // Remove all element that is not used during the exploration and repeated value
      
      new_q_lists_gpu.erase(thrust::remove_if(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), is_negative()),  new_q_lists_gpu.end() );
      thrust::sort(new_q_lists_gpu.begin(), new_q_lists_gpu.end());
      new_q_lists_gpu.erase(thrust::unique(new_q_lists_gpu.begin(), new_q_lists_gpu.end()), new_q_lists_gpu.end() );

      // std::cout << "new q size is" << new_q_lists_gpu.size() << std::endl;
      
      // Create new q list based on origional and updated q
      if (q_size <= max_thread_size) {
        q_lists_gpu.clear();
        q_lists_gpu = new_q_lists_gpu;
        new_q_lists_gpu.clear();
      }
      else {
        
        q_lists_gpu.erase(q_lists_gpu.begin(), q_lists_gpu.begin()+max_thread_size );
        q_lists_gpu.insert(q_lists_gpu.end(), new_q_lists_gpu.begin(), new_q_lists_gpu.end() );
        thrust::sort(q_lists_gpu.begin(), q_lists_gpu.end());
        q_lists_gpu.erase(thrust::unique(q_lists_gpu.begin(), q_lists_gpu.end()), q_lists_gpu.end() );
        new_q_lists_gpu.clear();

        // //sort the q_list based on the f value
        thrust::device_vector<float> f_value(q_lists_gpu.size());
        get_f<<<1, q_lists_gpu.size()>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(f_value.data()), q_lists_gpu.size() );
        thrust::sort_by_key(f_value.begin(), f_value.end(), q_lists_gpu.begin() );
      }

      
      //q_size = q_lists_gpu.size();
      // thrust::device_vector<float> h_value(q_size);

      // if (q_size > 1024) {
      //   int block = q_size / 1024 + 1;
        
      //   get_h<<<block, 1024>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(&h_value[0]), q_size );

      //   thrust::sort_by_key(h_value.begin(), h_value.end(), q_lists_gpu.begin() );

      // }

      if (path_found){
        while (path1 != start)
          {
              graph[path1].path = true;
              path1 = graph[path1].parent;
          }


      }

      

      std::vector<int32_t> v(n*n*n, 0);
      for (int z=0; z<n; z++){
            for (int y =0; y<n; y++){

                for (int x=0; x<n; x++){

                    if (graph[z*n*n + y*n+x].start) {
                        v[z*n*n + y*n+x] = 1;
                    }
                    else if (graph[z*n*n + y*n+x].goal)
                    {
                        v[z*n*n + y*n+x] = 2;
                    }
                    else if (graph[z*n*n + y*n+x].path){
                        v[z*n*n + y*n+x] = 3;
                    }
                    else if (graph[z*n*n + y*n+x].obstacle){
                        v[z*n*n + y*n+x] = 4;
                    }
                    else if (graph[z*n*n + y*n+x].frontier){
                        v[z*n*n + y*n+x] = 5;
                    }
                    else if (graph[z*n*n + y*n+x].explored){
                        v[z*n*n + y*n+x] = 6;
                    }
                    
                    
                }
            }
        }
      // thrust::host_vector<int> q_lists1= q_lists_gpu;
      // thrust::sort(q_lists1.begin(), q_lists1.end());
      // const bool hasDuplicates = std::adjacent_find(q_lists1.begin(), q_lists1.end()) != q_lists1.end();
      // std::cout << "Duplicates" << hasDuplicates << std::endl;
      
      // for (int k =0; k< q_lists1.size(); k++){

      //   v[q_lists1[k]] = 2;

      // }
      ros::Rate loop_rate(5);
            
      map.data = v;

      // map.points[10] = 120;
      // map.points[125] = 140;
  
  
      // 广播
      pub.publish(map);
      loop_rate.sleep(); 

      if (q_lists_gpu.size()==0) std::cout<<"NO PATH IS FOUND" <<std::endl;
            


    }
    
    ros::Rate loop_rate(5);
    pub.publish(map);
    loop_rate.sleep(); 

  }


  


  return 0;
}
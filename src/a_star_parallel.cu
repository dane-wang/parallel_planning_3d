#include "hip/hip_runtime.h"
#include <ros/ros.h> 
//#include <geometry_msgs>
// #include "geometry_msgs/Point.h"
#include <time.h>  
#include <vector>
#include "parallel_planning_3d/planner.h"
#include "std_msgs/Int8MultiArray.h"
#include <algorithm>
#include <xmlrpcpp/XmlRpcValue.h>

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/merge.h>
#include <queue>


__device__ bool path_found_gpu;
__device__ int neighbor_gpu[26];
__device__ int goal_gpu;


struct is_negative
{
  __host__ __device__
  bool operator()(int x)
  {
    return x ==-1;
  }
};



template <typename T, typename T1> 
__global__ void get_f(T* q,  planner::Node* graph, T1* h, int q_size )
{

  int tid = blockIdx.x *blockDim.x + threadIdx.x;
  if (tid < q_size){
    int node = q[tid];

    h[tid] = graph[node].f;

    // printf("%d", q[tid]);
  }

}

template <typename T>
__global__ void explore(T* q,  planner::Node* graph, T* new_q, int q_size  )
{
  int tid = blockIdx.x *blockDim.x + threadIdx.x;
  if (tid<q_size) {

    int explored_index = q[tid];
    int n = neighbor_gpu[2];

    int floor_index = explored_index%(n*n);
    int vertical_index = explored_index/(n*n);
    int row_index = floor_index / n;

    graph[explored_index].explored = true;
    graph[explored_index].frontier = false;

    if (graph[explored_index].goal){
      printf("FOUND");
      printf("Hello from thread %d, I am exploring %d\n", tid, explored_index);
      // planner::Node* temp_node = graph[explored_index].parent;
      // while (!temp_node->start){
        
      //     temp_node->path = true;
      //     temp_node = temp_node->parent;
      // }
      path_found_gpu = true;
    }

    if (!path_found_gpu){
      for (int i=0; i<26; i++)
      {   
        
        
        int new_index = explored_index + neighbor_gpu[i];
        float cost;
        
        if (i<6){
          cost = 1;
        }
        else if (i<18)
        {
          cost = sqrt(2.0);
        }
        else {
          cost = sqrt(3.0);
        }

        bool edge_detect = true;

        
                  
        bool left_edge_out = (floor_index%n ==0) && (i==1|| i==7 || i==9 || i==11 || i==15 || i==19 || i==21 || i==22 || i==24);

        bool right_edge_out = ((floor_index+1)%n ==0) && (i==0 || i==6 || i==8 || i==10 || i==14 || i==18 || i==20 || i==23 || i==25);

        bool front_edge_out = ((row_index+1)%n ==0) && (i==2 || i==6 || i==7 || i==12 || i==16 || i==18 || i==19 || i==24 || i==25);

        bool back_edge_out = (row_index == 0) && (i==3 || i==8 || i==9 || i==13 || i==17 || i==20 || i==21 || i==22 || i==23);

        bool top_edge_out = ((vertical_index+1)%n ==0) && (i==4 || i==10 || i==11 || i==12 || i==13 || i==20 || i==21 || i==18 || i==19);

        bool bot_edge_out = (vertical_index == 0) && (i==5 || i==14 || i==15 || i==16 || i==17 || i==24 || i==25 || i==22 || i==23);

        if (left_edge_out || right_edge_out || front_edge_out || back_edge_out || top_edge_out || bot_edge_out){
            edge_detect = false;
        }

        


        if (graph[new_index].obstacle == false && graph[new_index].frontier == false && graph[new_index].explored == false && edge_detect)
        {
          graph[new_index].g = graph[explored_index].g + cost;
            
          float h_1 = sqrt(pow((graph[new_index].x-graph[goal_gpu].x),2) + pow((graph[new_index].y-graph[goal_gpu].y),2) + pow((graph[new_index].z-graph[goal_gpu].z),2) );
            // printf("%f", h_1);
          graph[new_index].h = h_1;

            
          graph[new_index].f = graph[new_index].h + graph[new_index].g;
          graph[new_index].parent = explored_index;
          graph[new_index].frontier = true;
          
          new_q[26*tid+i] = new_index;
        }
        else if (edge_detect && graph[new_index].obstacle == false && (graph[new_index].frontier == true || graph[new_index].explored == true))
        {
          if (graph[new_index].g > graph[explored_index].g + cost)
          {
            graph[new_index].g = graph[explored_index].g + cost;
            graph[new_index].f = graph[new_index].h + graph[new_index].g;
            graph[new_index].parent = explored_index;
          }
        }
      }

    }
  }

}




  


int main(int argc, char** argv)
{
  ros::init(argc, argv, "parallel_planning");
  ros::NodeHandle nh; 

  // 发布消息 话题名字 队列大小
  ros::Publisher pub = nh.advertise<std_msgs::Int8MultiArray> ("planning_info", 100, ros::init_options::AnonymousName);
    
  //geometry_msgs::Point start_goal;
  std_msgs::Int8MultiArray map;
  

  //generate map info from the config file
  int n, max_thread_size;
  std::vector<int> start_coord, goal_coord;
  std::vector<int> obstacles;
  XmlRpc::XmlRpcValue xml_obstacles;

  ros::param::get("map_size", n);
  ros::param::get("start_position", start_coord);
  ros::param::get("goal_position", goal_coord);
  ros::param::get("obstacles", xml_obstacles);
  ros::param::get("max_thread", max_thread_size);

  // Initialize the start and goal node
  int start = start_coord[0]+ start_coord[1] * n + start_coord[2] * n * n;
  int goal = goal_coord[0] + goal_coord[1] * n + goal_coord[2] * n * n;


  // Initialize the obstacles list
  for(int i=0; i< xml_obstacles.size(); i++){
      int obstacles_index =  (int)xml_obstacles[i][0] +  (int)xml_obstacles[i][1] * n;
      obstacles.push_back( obstacles_index);
  }
  planner::Node graph[n*n*n];

  planner::map_generation(&graph[0], n, start, goal, obstacles);

  int path1 = goal;
  bool path_found = false;


  

  // Start to work with CUDA
  thrust::host_vector<int> q_lists;

  q_lists.push_back(start);

  // Start to allocate memory on gpu:
  
  const int map_size = n*n*n*sizeof(planner::Node);

  planner::Node *map_gpu;

  int neighbor[26] = {1, -1, n, -n, n*n, -n*n, n+1, n-1, -n+1, -n-1, n*n+1, n*n-1, n*n+n, n*n-n, -n*n+1, -n*n-1, -n*n+n, -n*n-n, n*n + n + 1, n*n + n- 1,  n*n - n + 1, n*n - n -1, -(n*n + n + 1), -(n*n + n- 1), -(n*n - n + 1), -(n*n - n -1) };

  hipMalloc( (void**)&map_gpu, map_size );
  hipMemcpy(map_gpu, &graph, map_size, hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(path_found_gpu), &path_found,  sizeof(bool));
  hipMemcpyToSymbol(HIP_SYMBOL(neighbor_gpu), &neighbor,  26*sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(goal_gpu), &goal,  sizeof(int));


  thrust::device_vector<int> q_lists_gpu = q_lists;

  // parallel_explore(&graph[0], n, path_found, start, max_thread_size);
  
  


  while (ros::ok()) {
    while(ros::ok() && q_lists_gpu.size()!=0 && !path_found){

      int q_size = q_lists_gpu.size();
      std::cout << "q size is" << q_size << std::endl;

      
      

      //Determine how many thread should be launched
      int thread_size_needed = min(max_thread_size, q_size);
      int block_size, thread_size;

      if (thread_size_needed <=1024){
        block_size = 1;
        thread_size = thread_size_needed;
      }
      else{
        block_size = (thread_size_needed/1024) + 1;
        thread_size = 1024;
      }

      //new_q is the list store the frontier generated from this step of exploration
      thrust::device_vector<int> new_q_lists_gpu(26*thread_size_needed);
      thrust::fill(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), -1);


 
      //Launch the kernel to explore the map
      explore<<<block_size,thread_size>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(new_q_lists_gpu.data()), q_size);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&path_found, HIP_SYMBOL(path_found_gpu),  sizeof(bool), 0, hipMemcpyDeviceToHost );

      
      hipMemcpy(&graph, map_gpu,  map_size, hipMemcpyDeviceToHost );


      // Remove all element that is not used during the exploration and repeated value
      
      new_q_lists_gpu.erase(thrust::remove_if(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), is_negative()),  new_q_lists_gpu.end() );
      
      new_q_lists_gpu.erase(thrust::unique(new_q_lists_gpu.begin(), new_q_lists_gpu.end()), new_q_lists_gpu.end() );
      
      // Create new q list based on origional and updated q
      if (q_size <= max_thread_size) {
        q_lists_gpu.clear();
        q_lists_gpu = new_q_lists_gpu;
        new_q_lists_gpu.clear();
      }
      else {
        
        q_lists_gpu.erase(q_lists_gpu.begin(), q_lists_gpu.begin()+max_thread_size );
        q_lists_gpu.insert(q_lists_gpu.end(), new_q_lists_gpu.begin(), new_q_lists_gpu.end() );
        new_q_lists_gpu.clear();

        // //sort the q_list based on the f value
        thrust::device_vector<float> f_value(q_lists_gpu.size());
        get_f<<<1, q_lists_gpu.size()>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(f_value.data()), q_lists_gpu.size() );
        thrust::sort_by_key(f_value.begin(), f_value.end(), q_lists_gpu.begin() );
      }

      
      //q_size = q_lists_gpu.size();
      // thrust::device_vector<float> h_value(q_size);

      // if (q_size > 1024) {
      //   int block = q_size / 1024 + 1;
        
      //   get_h<<<block, 1024>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(&h_value[0]), q_size );

      //   thrust::sort_by_key(h_value.begin(), h_value.end(), q_lists_gpu.begin() );

      // }

      if (path_found){
        while (path1 != start)
          {
              graph[path1].path = true;
              path1 = graph[path1].parent;
          }


      }

      

      std::vector<int8_t> v(n*n*n, 0);
      for (int z=0; z<n; z++){
            for (int y =0; y<n; y++){

                for (int x=0; x<n; x++){

                    if (graph[z*n*n + y*n+x].start) {
                        v[z*n*n + y*n+x] = 1;
                    }
                    else if (graph[z*n*n + y*n+x].goal)
                    {
                        v[z*n*n + y*n+x] = 2;
                    }
                    else if (graph[z*n*n + y*n+x].path){
                        v[z*n*n + y*n+x] = 3;
                    }
                    else if (graph[z*n*n + y*n+x].obstacle){
                        v[z*n*n + y*n+x] = 4;
                    }
                    else if (graph[z*n*n + y*n+x].frontier){
                        v[z*n*n + y*n+x] = 5;
                    }
                    else if (graph[z*n*n + y*n+x].explored){
                        v[z*n*n + y*n+x] = 6;
                    }
                    
                    
                }
            }
        }
      
      // for (int k =0; k< n*n; k++){

      //   std::cout<< static_cast<int16_t>(v[k]) << std::endl;

      // }
      ros::Rate loop_rate(5);
            
      map.data = v;

      // map.points[10] = 120;
      // map.points[125] = 140;
  
  
      // 广播
      pub.publish(map);
      loop_rate.sleep(); 

    }
    if (path_found){
      ros::Rate loop_rate(5);
      pub.publish(map);
      loop_rate.sleep(); 

  }

  }


  


  return 0;
}
#include "hip/hip_runtime.h"
#include <ros/ros.h> 
//#include <geometry_msgs>
// #include "geometry_msgs/Point.h"
#include <time.h>  
#include <vector>
#include "parallel_planning_3d/planner.h"
#include "std_msgs/Int8MultiArray.h"
#include <algorithm>
#include <xmlrpcpp/XmlRpcValue.h>

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/merge.h>
#include <queue>
#include <random>


__device__ bool path_found_gpu;
__device__ int neighbor_gpu[26];
__device__ int goal_gpu;
__device__ int start_gpu;


struct is_negative
{
  __host__ __device__
  bool operator()(int x)
  {
    return x ==-1;
  }
};



template <typename T, typename T1> 
__global__ void get_f(T* q,  planner::BiNode* graph, T1* h, int q_size )
{

  int tid = blockIdx.x *blockDim.x + threadIdx.x;
  if (tid < q_size){
    int node = q[tid];

    h[tid] = graph[node].f;

    // printf("%d", q[tid]);
  }

}

template <typename T, typename T1> 
__global__ void get_b_f(T* b_q,  planner::BiNode* graph, T1* h, int q_size )
{

  int tid = blockIdx.x *blockDim.x + threadIdx.x;
  if (tid < q_size){
    int node = b_q[tid];

    h[tid] = graph[node].b_f;

    // printf("%d", q[tid]);
  }

}

template <typename T, typename T1> 
__global__ void get_total_g(T* meeting_node,  planner::BiNode* graph, T1* h, int meeting_size )
{

  int tid = blockIdx.x *blockDim.x + threadIdx.x;
  if (tid < meeting_size){
    int node = meeting_node[tid];

    h[tid] = graph[node].b_g + graph[node].g ;

    // printf("%d", q[tid]);
  }

}




template <typename T>
__global__ void explore(planner::BiNode* graph, T* q,  T* new_q, T* b_q, T* new_b_q, T* meeting_nodes, int q_thread, int b_q_thread  )
{
  int tid = blockIdx.x *blockDim.x + threadIdx.x;

  if (tid<q_thread) {

    int explored_index = q[tid];
    int n = neighbor_gpu[2];

    int floor_index = explored_index%(n*n);
    int vertical_index = explored_index/(n*n);
    int row_index = floor_index / n;

    graph[explored_index].explored = true;
    graph[explored_index].frontier = false;

    if (graph[explored_index].b_explored){
      printf("FOUND");
      // printf("Hello from thread %d, I am exploring %d\n", tid, explored_index);
      // planner::Node* temp_node = graph[explored_index].parent;
      // while (!temp_node->start){
        
      //     temp_node->path = true;
      //     temp_node = temp_node->parent;
      // }
      meeting_nodes[tid] = explored_index;
      path_found_gpu = true;
    }

    if (!path_found_gpu){
      for (int i=0; i<26; i++)
      {   
        
        
        int new_index = explored_index + neighbor_gpu[i];
        
        if (new_index<0 || new_index >= n*n*n) continue;

        float cost;
        
        if (i<6){
          cost = 1;
        }
        else if (i<18)
        {
          cost = sqrt(2.0);
        }
        else {
          cost = sqrt(3.0);
        }

        bool edge_detect = true;

        
                  
        bool left_edge_out = (floor_index%n ==0) && (i==1|| i==7 || i==9 || i==11 || i==15 || i==19 || i==21 || i==22 || i==24);

        bool right_edge_out = ((floor_index+1)%n ==0) && (i==0 || i==6 || i==8 || i==10 || i==14 || i==18 || i==20 || i==23 || i==25);

        bool front_edge_out = ((row_index+1)%n ==0) && (i==2 || i==6 || i==7 || i==12 || i==16 || i==18 || i==19 || i==24 || i==25);

        bool back_edge_out = (row_index == 0) && (i==3 || i==8 || i==9 || i==13 || i==17 || i==20 || i==21 || i==22 || i==23);

        bool top_edge_out = ((vertical_index+1)%n ==0) && (i==4 || i==10 || i==11 || i==12 || i==13 || i==20 || i==21 || i==18 || i==19);

        bool bot_edge_out = (vertical_index == 0) && (i==5 || i==14 || i==15 || i==16 || i==17 || i==24 || i==25 || i==22 || i==23);

        if (left_edge_out || right_edge_out || front_edge_out || back_edge_out || top_edge_out || bot_edge_out){
            edge_detect = false;
        }

        


        if (graph[new_index].obstacle == false && graph[new_index].frontier == false && graph[new_index].explored == false && edge_detect)
        {
          graph[new_index].g = graph[explored_index].g + cost;
            
          float h_1 = sqrt(pow((graph[new_index].x-graph[goal_gpu].x),2) + pow((graph[new_index].y-graph[goal_gpu].y),2) + pow((graph[new_index].z-graph[goal_gpu].z),2) );
            // printf("%f", h_1);
          graph[new_index].h = h_1;

            
          graph[new_index].f = graph[new_index].h + graph[new_index].g;
          graph[new_index].parent = explored_index;
          graph[new_index].frontier = true;
          
          new_q[26*tid+i] = new_index;
        }
        else if (edge_detect && graph[new_index].obstacle == false && (graph[new_index].frontier == true || graph[new_index].explored == true))
        {
          if (graph[new_index].g > graph[explored_index].g + cost)
          {
            graph[new_index].g = graph[explored_index].g + cost;
            graph[new_index].f = graph[new_index].h + graph[new_index].g;
            graph[new_index].parent = explored_index;
          }
        }
      }

    }
  }
  else if (tid<(q_thread+b_q_thread))
  {
    int explored_index = b_q[tid-q_thread];
    int n = neighbor_gpu[2];

    int floor_index = explored_index%(n*n);
    int vertical_index = explored_index/(n*n);
    int row_index = floor_index / n;

    graph[explored_index].b_explored = true;
    graph[explored_index].b_frontier = false;

    if (graph[explored_index].explored){
      printf("FOUND");
      // printf("Hello from thread %d, I am exploring %d\n", tid, explored_index);
      // planner::Node* temp_node = graph[explored_index].parent;
      // while (!temp_node->start){
        
      //     temp_node->path = true;
      //     temp_node = temp_node->parent;
      // }
      meeting_nodes[tid] = explored_index;
      path_found_gpu = true;
    }

    if (!path_found_gpu){
      for (int i=0; i<26; i++)
      {   
        
        
        int new_index = explored_index + neighbor_gpu[i];
        
        if (new_index<0 || new_index >= n*n*n) continue;

        float cost;
        
        if (i<6){
          cost = 1;
        }
        else if (i<18)
        {
          cost = sqrt(2.0);
        }
        else {
          cost = sqrt(3.0);
        }

        bool edge_detect = true;

        
                  
        bool left_edge_out = (floor_index%n ==0) && (i==1|| i==7 || i==9 || i==11 || i==15 || i==19 || i==21 || i==22 || i==24);

        bool right_edge_out = ((floor_index+1)%n ==0) && (i==0 || i==6 || i==8 || i==10 || i==14 || i==18 || i==20 || i==23 || i==25);

        bool front_edge_out = ((row_index+1)%n ==0) && (i==2 || i==6 || i==7 || i==12 || i==16 || i==18 || i==19 || i==24 || i==25);

        bool back_edge_out = (row_index == 0) && (i==3 || i==8 || i==9 || i==13 || i==17 || i==20 || i==21 || i==22 || i==23);

        bool top_edge_out = ((vertical_index+1)%n ==0) && (i==4 || i==10 || i==11 || i==12 || i==13 || i==20 || i==21 || i==18 || i==19);

        bool bot_edge_out = (vertical_index == 0) && (i==5 || i==14 || i==15 || i==16 || i==17 || i==24 || i==25 || i==22 || i==23);

        if (left_edge_out || right_edge_out || front_edge_out || back_edge_out || top_edge_out || bot_edge_out){
            edge_detect = false;
        }

        


        if (graph[new_index].obstacle == false && graph[new_index].b_frontier == false && graph[new_index].b_explored == false && edge_detect)
        {
          graph[new_index].b_g = graph[explored_index].b_g + cost;
            
          float h_1 = sqrt(pow((graph[new_index].x-graph[start_gpu].x),2) + pow((graph[new_index].y-graph[start_gpu].y),2) + pow((graph[new_index].z-graph[start_gpu].z),2) );
            // printf("%f", h_1);
          graph[new_index].b_h = h_1;

            
          graph[new_index].b_f = graph[new_index].b_h + graph[new_index].b_g;
          graph[new_index].b_parent = explored_index;
          graph[new_index].b_frontier = true;
          
          new_b_q[26*(tid-q_thread)+i] = new_index;
        }
        else if (edge_detect && graph[new_index].obstacle == false && (graph[new_index].b_frontier == true || graph[new_index].b_explored == true))
        {
          if (graph[new_index].b_g > graph[explored_index].b_g + cost)
          {
            graph[new_index].b_g = graph[explored_index].b_g + cost;
            graph[new_index].b_f = graph[new_index].b_h + graph[new_index].b_g;
            graph[new_index].b_parent = explored_index;
          }
        }
      }
    }

  }
  

}




  


int main(int argc, char** argv)
{
    
    ros::init(argc, argv, "parallel_planning");
    ros::NodeHandle nh; 

    // 发布消息 话题名字 队列大小
    ros::Publisher pub = nh.advertise<std_msgs::Int8MultiArray> ("planning_info", 100, ros::init_options::AnonymousName);
        
    //geometry_msgs::Point start_goal;
    std_msgs::Int8MultiArray map;
    

    //generate map info from the config file
    int n, max_thread_size, use_random_obstacles;
    std::vector<int> start_coord, goal_coord;
    std::vector<int> obstacles;
    

    ros::param::get("map_size", n);
    ros::param::get("start_position", start_coord);
    ros::param::get("goal_position", goal_coord);
    ros::param::get("use_random_obstacles", use_random_obstacles);
    ros::param::get("max_thread", max_thread_size);

    // Initialize the start and goal node
    int start = start_coord[0]+ start_coord[1] * n + start_coord[2] * n * n;
    int goal = goal_coord[0] + goal_coord[1] * n + goal_coord[2] * n * n;

    if(use_random_obstacles){

            float ratio;
            ros::param::get("random_obstacles_ratio", ratio);

            int obstacle_size = ratio * n;

            // std::cout<<"obstacle "<< obstacle_size<< std::endl;

            // First create an instance of an engine.
            std::random_device rnd_device;
            // Specify the engine and distribution.
            std::mt19937 mersenne_engine {rnd_device()};  // Generates random integers
            std::uniform_int_distribution<int> dist {0, n*n*n-1};
            
            auto gen = [&dist, &mersenne_engine](){
                        return dist(mersenne_engine);
                    };

            
            std::vector<int> vec(obstacle_size);
            std::generate(std::begin(vec), std::end(vec), gen);
            obstacles = vec;

            

        }
        else{

            XmlRpc::XmlRpcValue xml_obstacles;
            ros::param::get("obstacles", xml_obstacles);
            for(int i=0; i< xml_obstacles.size(); i++){
                int obstacles_index =  (int)xml_obstacles[i][0] +  (int)xml_obstacles[i][1] * n + (int)xml_obstacles[i][2] * n * n;
                obstacles.push_back(obstacles_index);
            }
    }


    auto graph = new planner::BiNode[n*n*n];

    planner::map_generation(graph, n, start, goal, obstacles);

    
    bool path_found = false;

    // Start to work with CUDA
    thrust::host_vector<int> q_lists;
    thrust::host_vector<int> b_q_lists;

    q_lists.push_back(start);
    b_q_lists.push_back(goal);

    

    // Start to allocate memory on gpu:
    
    const int map_size = n*n*n*sizeof(planner::BiNode);

    planner::BiNode *map_gpu;

    int neighbor[26] = {1, -1, n, -n, n*n, -n*n, n+1, n-1, -n+1, -n-1, n*n+1, n*n-1, n*n+n, n*n-n, -n*n+1, -n*n-1, -n*n+n, -n*n-n, n*n + n + 1, n*n + n- 1,  n*n - n + 1, n*n - n -1, -(n*n + n + 1), -(n*n + n- 1), -(n*n - n + 1), -(n*n - n -1) };

    hipMalloc( (void**)&map_gpu, map_size );
    hipMemcpy(map_gpu, graph, map_size, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(path_found_gpu), &path_found,  sizeof(bool));
    hipMemcpyToSymbol(HIP_SYMBOL(neighbor_gpu), &neighbor,  26*sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(goal_gpu), &goal,  sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(start_gpu), &start,  sizeof(int));


    thrust::device_vector<int> q_lists_gpu = q_lists;
    thrust::device_vector<int> b_q_lists_gpu = b_q_lists;

 
    


    while (ros::ok()) {
        // std::cout << "q size is" << q_lists_gpu.size() << std::endl;
        // std::cout << "b q size is" << b_q_lists_gpu.size() << std::endl;

        while(ros::ok() && q_lists_gpu.size()!=0 && b_q_lists_gpu.size()!=0  && !path_found){

        int q_size = q_lists_gpu.size();
        int b_q_size = b_q_lists_gpu.size();
        // std::cout << "q size is" << q_size << std::endl;
        // std::cout << "b q size is" << b_q_size << std::endl;

        
        

        //Determine how many thread should be launched
        int thread_size_needed = min(max_thread_size, (q_size+b_q_size));
        int block_size, thread_size;
        int q_thread, b_q_thread;

        if (thread_size_needed <=1024){
            block_size = 1;
            thread_size = thread_size_needed;
        }
        else{
            block_size = (thread_size_needed/1024) + 1;
            thread_size = 1024;
        }

        //new_q is the list store the frontier generated from this step of exploration
        if ((q_size+b_q_size)<=max_thread_size){

            q_thread = q_size;
            b_q_thread = b_q_size;

        }
        else{

            float ratio = q_size / (q_size+b_q_size);
            q_thread = (int) (ratio * max_thread_size);
            q_thread = max(q_thread, 300);
            b_q_thread = max_thread_size - q_thread;

            if (b_q_thread > b_q_size) {
              b_q_thread = b_q_size;
              q_thread = max_thread_size - b_q_thread;
            }


        }

        std::cout << "q size is" << q_size << std::endl;
        std::cout << "b q size is" << b_q_size << std::endl;

        std::cout << "q thread is" << q_thread << std::endl;
        std::cout << "b q thread is" << b_q_thread << std::endl;

        thrust::device_vector<int> new_q_lists_gpu(q_thread*26);
        thrust::fill(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), -1);

        thrust::device_vector<int> new_b_q_lists_gpu(b_q_thread*26);
        thrust::fill(new_b_q_lists_gpu.begin(), new_b_q_lists_gpu.end(), -1);

        thrust::device_vector<int> meeting_nodes(q_thread + b_q_thread);
        thrust::fill(meeting_nodes.begin(), meeting_nodes.end(), -1);
    


    
        //Launch the kernel to explore the map
        explore<<<block_size,thread_size>>>(map_gpu, thrust::raw_pointer_cast(q_lists_gpu.data()), thrust::raw_pointer_cast(new_q_lists_gpu.data()),  thrust::raw_pointer_cast(b_q_lists_gpu.data()), thrust::raw_pointer_cast(new_b_q_lists_gpu.data()), thrust::raw_pointer_cast(meeting_nodes.data()), q_thread, b_q_thread);
        hipDeviceSynchronize();
        hipMemcpyFromSymbol(&path_found, HIP_SYMBOL(path_found_gpu),  sizeof(bool), 0, hipMemcpyDeviceToHost );

        
        hipMemcpy(graph, map_gpu,  map_size, hipMemcpyDeviceToHost );


        // Remove all element that is not used during the exploration and repeated value
        
        new_q_lists_gpu.erase(thrust::remove_if(new_q_lists_gpu.begin(), new_q_lists_gpu.end(), is_negative()),  new_q_lists_gpu.end() );
        thrust::sort(new_q_lists_gpu.begin(), new_q_lists_gpu.end());
        new_q_lists_gpu.erase(thrust::unique(new_q_lists_gpu.begin(), new_q_lists_gpu.end()), new_q_lists_gpu.end() );

        new_b_q_lists_gpu.erase(thrust::remove_if(new_b_q_lists_gpu.begin(), new_b_q_lists_gpu.end(), is_negative()),  new_b_q_lists_gpu.end() );
        thrust::sort(new_b_q_lists_gpu.begin(), new_b_q_lists_gpu.end());
        new_b_q_lists_gpu.erase(thrust::unique(new_b_q_lists_gpu.begin(), new_b_q_lists_gpu.end()), new_b_q_lists_gpu.end() );


        // std::cout << "new q size is" << new_q_lists_gpu.size() << std::endl;
        
        // Create new q list based on origional and updated q
        if (q_size <= q_thread) {
            q_lists_gpu.clear();
            q_lists_gpu = new_q_lists_gpu;
            new_q_lists_gpu.clear();
        }
        else {
            
            q_lists_gpu.erase(q_lists_gpu.begin(), q_lists_gpu.begin()+q_thread );
            q_lists_gpu.insert(q_lists_gpu.end(), new_q_lists_gpu.begin(), new_q_lists_gpu.end() );
            thrust::sort(q_lists_gpu.begin(), q_lists_gpu.end());
            q_lists_gpu.erase(thrust::unique(q_lists_gpu.begin(), q_lists_gpu.end()), q_lists_gpu.end() );
            new_q_lists_gpu.clear();

            // //sort the q_list based on the f value
            thrust::device_vector<float> f_value(q_lists_gpu.size());
            get_f<<<1, q_lists_gpu.size()>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(f_value.data()), q_lists_gpu.size() );
            thrust::sort_by_key(f_value.begin(), f_value.end(), q_lists_gpu.begin() );
            // thrust::reverse(thrust::device, q_lists_gpu.begin(), q_lists_gpu.end());
            
        }

        if (b_q_size <= b_q_thread) {
            b_q_lists_gpu.clear();
            b_q_lists_gpu = new_b_q_lists_gpu;
            new_b_q_lists_gpu.clear();
        }
        else {
            
            b_q_lists_gpu.erase(b_q_lists_gpu.begin(), b_q_lists_gpu.begin()+b_q_thread );
            b_q_lists_gpu.insert(b_q_lists_gpu.end(), new_b_q_lists_gpu.begin(), new_b_q_lists_gpu.end() );
            thrust::sort(b_q_lists_gpu.begin(), b_q_lists_gpu.end());
            b_q_lists_gpu.erase(thrust::unique(b_q_lists_gpu.begin(), b_q_lists_gpu.end()), b_q_lists_gpu.end() );
            new_b_q_lists_gpu.clear();

            // //sort the q_list based on the f value
            thrust::device_vector<float> b_f_value(b_q_lists_gpu.size());
            get_b_f<<<1, b_q_lists_gpu.size()>>>(thrust::raw_pointer_cast(b_q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(b_f_value.data()), b_q_lists_gpu.size() );
            thrust::sort_by_key(b_f_value.begin(), b_f_value.end(), b_q_lists_gpu.begin() );
            // thrust::reverse(thrust::device, b_q_lists_gpu.begin(), b_q_lists_gpu.end());
            
        }

        
        //q_size = q_lists_gpu.size();
        // thrust::device_vector<float> h_value(q_size);

        // if (q_size > 1024) {
        //   int block = q_size / 1024 + 1;
            
        //   get_h<<<block, 1024>>>(thrust::raw_pointer_cast(q_lists_gpu.data()),  map_gpu, thrust::raw_pointer_cast(&h_value[0]), q_size );

        //   thrust::sort_by_key(h_value.begin(), h_value.end(), q_lists_gpu.begin() );

        // }

        if (path_found){

          std::cout<< "checking for path" << std::endl;

          std::cout<< "Possible middle point number before remove" << meeting_nodes.size() << std::endl;
          meeting_nodes.erase(thrust::remove_if(meeting_nodes.begin(), meeting_nodes.end(), is_negative()),  meeting_nodes.end() );
          thrust::sort(meeting_nodes.begin(), meeting_nodes.end());
          meeting_nodes.erase(thrust::unique(meeting_nodes.begin(), meeting_nodes.end()), meeting_nodes.end() );

          
          std::cout<< "Possible middle point number" << meeting_nodes.size() << std::endl;
          
          thrust::device_vector<float> total_g_value(meeting_nodes.size());
          get_total_g<<<1, meeting_nodes.size()>>>(thrust::raw_pointer_cast(meeting_nodes.data()),  map_gpu, thrust::raw_pointer_cast(total_g_value.data()), meeting_nodes.size() );

          auto it = std::min_element(std::begin(total_g_value), std::end(total_g_value));

          int middle_point = meeting_nodes[std::distance(std::begin(total_g_value), it)];

          int path1 = middle_point, path2 = middle_point;

          while (path1 != start)
          {
              graph[path1].path = true;
              path1 = graph[path1].parent;
          }

          while (path2 != goal)
          {
              graph[path2].path = true;
              path2 = graph[path2].b_parent;
          }


        }

        

        std::vector<int8_t> v(n*n*n, 0);
        for (int z=0; z<n; z++){
                for (int y =0; y<n; y++){

                    for (int x=0; x<n; x++){

                        if (graph[z*n*n + y*n+x].start) {
                            v[z*n*n + y*n+x] = 1;
                        }
                        else if (graph[z*n*n + y*n+x].goal)
                        {
                            v[z*n*n + y*n+x] = 2;
                        }
                        else if (graph[z*n*n + y*n+x].path){
                            v[z*n*n + y*n+x] = 3;
                        }
                        else if (graph[z*n*n + y*n+x].obstacle){
                            v[z*n*n + y*n+x] = 4;
                        }
                        else if (graph[z*n*n + y*n+x].frontier || graph[z*n*n + y*n+x].b_frontier){
                            v[z*n*n + y*n+x] = 5;
                        }
                        else if (graph[z*n*n + y*n+x].explored){
                            v[z*n*n + y*n+x] = 6;
                        }
                        else if (graph[z*n*n + y*n+x].b_explored){
                            v[z*n*n + y*n+x] = 7;
                        }
                        
                        
                    }
                }
            }
        // thrust::host_vector<int> q_lists1= q_lists_gpu;
        // thrust::sort(q_lists1.begin(), q_lists1.end());
        // const bool hasDuplicates = std::adjacent_find(q_lists1.begin(), q_lists1.end()) != q_lists1.end();
        // std::cout << "Duplicates" << hasDuplicates << std::endl;
        
        // for (int k =0; k< q_lists1.size(); k++){

        //   v[q_lists1[k]] = 2;

        // }
        ros::Rate loop_rate(5);
                
        map.data = v;

        // map.points[10] = 120;
        // map.points[125] = 140;
    
    
        // 广播
        pub.publish(map);
        loop_rate.sleep(); 

        if (q_lists_gpu.size()==0) std::cout<<"NO PATH IS FOUND" <<std::endl;
                


        }
        
        ros::Rate loop_rate(5);
        pub.publish(map);
        loop_rate.sleep(); 

    }


    


    return 0;
    }